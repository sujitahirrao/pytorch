#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorFactories.h>
#include <ATen/cuda/CubUtils.cuh>

#include <limits>

namespace at {
namespace native {

Tensor& randperm_out_cuda(int64_t n, c10::optional<Generator> generator, Tensor& result) {
  TORCH_CHECK(n >= 0, "n must be non-negative, got", n);
  TORCH_CHECK(!generator.has_value() || (generator.has_value() && result.device() == generator->device()), "Expected a '", result.device(), "' generator device but found '", generator->device(), "'");
  check_supported_max_int_with_precision(n, result);

  result.resize_({n});

  if (n < 30000) {  // For small inputs, we offload it to CPU instead.
    auto result_cpu = at::empty({n}, result.options().device(kCPU));
    randperm_out(result_cpu, n, generator);
    return result.copy_(result_cpu);
  }

#if 0
  // This if condition should never be true because if n >= 30000 and the tensor has a Half type,
  // check_supported_max_int_with_precision should have reported an error. This snippet is commented out but left here
  // for the sake of clarity, because Half in thrust is spotty, and we do not want future change unaware of this.
  if (result.scalar_type() == at::ScalarType::Half) {  // Half in thrust is spotty. Avoid.
    auto result_float = at::empty({n}, initialTensorOptions().device(Device(DeviceType::CUDA)));
    return result.copy_(randperm_out_cuda(result_float, n, generator));
  }
#endif

  // Generate random values for the keys array
  AT_DISPATCH_ALL_TYPES(
    result.scalar_type(), "randperm_out_cuda", [&] {
      TORCH_CHECK(n <= std::numeric_limits<int>::max(),
        "randperm of tensors larger than INT_MAX is not supported yet in pytorch");

      auto keys = at::empty(result.sizes(), result.options()).random_(generator);
      auto range = at::arange(n, result.options());
      auto keys_tmp = at::empty_like(keys);

      // shuffled_data points to the underlying data of the output tensor if the tensor is contiguous; otherwise it
      // points to a new tensor.
      Tensor shuffled;
      scalar_t *shuffled_data;
      if (result.is_contiguous()) {
        shuffled_data = result.data_ptr<scalar_t>();
      } else {
        shuffled = at::empty(n, result.options());
        shuffled_data = shuffled.data_ptr<scalar_t>();
      }

      // Use the sorted order of keys to rearrange the result array
      size_t temp_storage_bytes = 0;

      hipcub::DeviceRadixSort::SortPairs(
        nullptr, temp_storage_bytes,
        keys.data_ptr<scalar_t>(), keys_tmp.data_ptr<scalar_t>(),
        range.data_ptr<scalar_t>(), shuffled_data, n,
        0, sizeof(scalar_t) * 8, at::cuda::getCurrentCUDAStream());
      auto& allocator = *::c10::cuda::CUDACachingAllocator::get();
      auto dataPtr = allocator.allocate(temp_storage_bytes);
      hipcub::DeviceRadixSort::SortPairs(
        dataPtr.get(), temp_storage_bytes,
        keys.data_ptr<scalar_t>(), keys_tmp.data_ptr<scalar_t>(),
        range.data_ptr<scalar_t>(), shuffled_data, n,
        0, sizeof(scalar_t) * 8, at::cuda::getCurrentCUDAStream());

      if (!result.is_contiguous()) {
        result.copy_(shuffled);
      }
    }
  );

  return result;
}



}} // namespace at::native
