#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/PinnedMemoryAllocator.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDASolver.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/LinearAlgebraUtils.h>
#include <ATen/native/cuda/MiscUtils.h>
#include <ATen/native/cuda/BatchLinearAlgebraLib.h>

#ifdef USE_CUSOLVER

namespace at {
namespace native {

inline static Tensor column_major_identity_matrix_like(const Tensor& self) {
  auto size = self.sizes();
  auto size_slice = IntArrayRef(size.data(), size.size()-1);
  return at::ones(size_slice, self.options()).diag_embed().transpose(-2, -1);
}

template <typename scalar_t>
inline static void _apply_single_inverse_helper(scalar_t* self_ptr, scalar_t* self_inv_ptr, int* ipiv_ptr, int* info_getrf_ptr, int* info_getrs_ptr, int n, int lda) {
  // self_inv_ptr should already be an identity matrix

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  at::cuda::solver::getrf<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, info_getrf_ptr);
  at::cuda::solver::getrs<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, self_inv_ptr, lda, info_getrs_ptr);
}

template <typename scalar_t>
static void apply_batched_inverse_lib(Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");
  const int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  const int lda = std::max<int>(1, n);

  auto self_data = self.data_ptr<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data_ptr<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  auto infos_getrf_data = infos_getrf.data_ptr<int>();
  auto infos_getrs_data = infos_getrs.data_ptr<int>();

  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

  if (use_loop_launch(batch_size, n)) {
    auto main_stream = at::cuda::getCurrentCUDAStream();

    at::cuda::CUDAEvent main_event;
    main_event.record(main_stream);

    for (int64_t i = 0; i < batch_size; i++) {
      auto stream = at::cuda::getStreamFromPool();
      at::cuda::CUDAStreamGuard guard(stream);

      main_event.block(stream);

      auto dataPtr = allocator.allocate(sizeof(int) * lda);
      int* pivot = reinterpret_cast<int*>(dataPtr.get());

      int* infos_getrf_working_ptr = &infos_getrf_data[i];
      int* infos_getrs_working_ptr = &infos_getrs_data[i];

      _apply_single_inverse_helper<scalar_t>(
        &self_data[i * self_mat_stride], &self_inv_data[i * self_inv_mat_stride], pivot, infos_getrf_working_ptr, infos_getrs_working_ptr, n, lda);

      at::cuda::CUDAEvent finished;
      finished.record(stream);
      finished.block(main_stream);
    }
  } else {
    // cublas batched kernels require input be "device array of device pointers"
    Tensor self_array = at::arange(
      reinterpret_cast<long>(self_data),
      reinterpret_cast<long>(&self_data[(batch_size-1) * self_mat_stride]) + 1,
      static_cast<long>(self_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));
    Tensor self_inv_array = at::arange(
      reinterpret_cast<long>(self_inv_data),
      reinterpret_cast<long>(&self_inv_data[(batch_size-1) * self_inv_mat_stride]) + 1,
      static_cast<long>(self_inv_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));

    auto dataPtr = allocator.allocate(sizeof(int)*batch_size*lda);
    int* ipiv_array = reinterpret_cast<int*>(dataPtr.get());

    at::cuda::blas::getrfBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, infos_getrf_data, batch_size);

    at::cuda::blas::getriBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, reinterpret_cast<scalar_t**>(self_inv_array.data_ptr()), lda, infos_getrs_data, batch_size);
  }
}

template <typename scalar_t>
static void apply_single_inverse_lib(const Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  int lda = std::max<int>(1, n);

  Tensor ipiv = at::empty({lda}, self.options().dtype(at::kInt));

  _apply_single_inverse_helper<scalar_t>(
    self.data_ptr<scalar_t>(), self_inv.data_ptr<scalar_t>(), ipiv.data_ptr<int>(), infos_getrf.data_ptr<int>(), infos_getrs.data_ptr<int>(), n, lda);
}

// This is a type dispatching helper function for 'apply_batched_inverse_lib' and 'apply_single_inverse_lib'
Tensor& _linalg_inv_out_helper_cuda_lib(Tensor& result, Tensor& infos_getrf, Tensor& infos_getrs) {
  // assuming result is in column major order and contains the matrices to invert
  Tensor input_working_copy = cloneBatchedColumnMajor(result);

  // for getrf + getrs (cusolver path)
  // result should be filled with identity matrices
  result.zero_();
  result.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);

  const int batch_size = cuda_int_cast(batchCount(result), "batchCount");

  if (result.dim() > 2) {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        input_working_copy, result, infos_getrf, infos_getrs);
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(input_working_copy, result, infos_getrf, infos_getrs);
    });
  }

  return result;
}

Tensor _inverse_helper_cuda_lib(const Tensor& self) {
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  Tensor self_inv_working_copy = column_major_identity_matrix_like(self_working_copy);
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");

  if (self.dim() > 2 && batch_size > 1) {
    Tensor infos_getrf = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  } else {
    Tensor infos_getrf = at::zeros({1}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({1}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  }

  return self_inv_working_copy;
}

}} // namespace at::native

#endif  // USE_CUSOLVER
