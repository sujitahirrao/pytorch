#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Config.h>
#include <ATen/Dispatch.h>
#include <ATen/Utils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/detail/CUDAHooksInterface.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/SpectralOpsUtils.h>
#include <ATen/native/cuda/CuFFTUtils.h>
#include <ATen/native/cuda/CuFFTPlanCache.h>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <vector>
#include <cmath>

namespace at { namespace native {

using namespace at::native::detail;

// Offset calculator for indexing in Hermitian mirrored order.
// In mirrored dims, maps linear index i to (n - i) % n
template <typename index_t>
struct HermitianSymmetryOffsetCalculator {
  using offset_type = at::detail::Array<index_t, 1>;
  using dim_type = std::remove_cv_t<decltype(MAX_DIMS)>;
  dim_type dims;
  IntDivider<index_t> sizes_[MAX_DIMS];
  index_t strides_[MAX_DIMS];
  uint32_t mirror_dim_;  // bit mask
  static_assert(MAX_DIMS < 32, "Need a bigger mask type");

  HermitianSymmetryOffsetCalculator(
      IntArrayRef sizes, IntArrayRef strides, IntArrayRef dim,
      const int64_t element_size){
    TORCH_INTERNAL_ASSERT(sizes.size() == strides.size());
    TORCH_INTERNAL_ASSERT(sizes.size() <= MAX_DIMS);
    dims = sizes.size();

    for (dim_type i = 0; i < MAX_DIMS; ++i) {
      if (i < dims) {
        sizes_[i] = IntDivider<index_t>(sizes[i]);
        strides_[i] = strides[i] / element_size;
      } else {
        sizes_[i] = IntDivider<index_t>(1);
        strides_[i] = 0;
      }
    }

    mirror_dim_ = 0;
    for (int64_t i = 0; i < dim.size(); ++i) {
      mirror_dim_ |= (uint32_t{1} << dim[i]);
    }
  }

  C10_HOST_DEVICE offset_type get(index_t linear_idx) const {
    index_t offset = 0;

    for (dim_type dim = 0; dim < dims; ++dim) {
      auto divmod = sizes_[dim].divmod(linear_idx);
      linear_idx = divmod.div;

      if ((mirror_dim_ & (uint32_t{1} << dim)) == 0) {
        offset += divmod.mod * strides_[dim];
      } else if (divmod.mod != 0) {
        offset += (sizes_[dim].divisor - divmod.mod) * strides_[dim];
      }
    }
    offset_type offsets;
    offsets[0] = offset;
    return offsets;
  }
};

// out[:] = conj(in[:]) where in and out ordering is generalized by offset calculators
template <typename scalar_t, typename inp_calc_t, typename out_calc_t>
C10_LAUNCH_BOUNDS_1(cuda::detail::CUDA_NUM_THREADS)
__global__ void _fft_conjugate_copy_kernel(
    int64_t numel, scalar_t * out_data, const scalar_t * in_data,
    inp_calc_t ic, out_calc_t oc) {
  CUDA_KERNEL_LOOP_TYPE(index, numel, int64_t) {
    auto in_offset = ic.get(index)[0];
    auto out_offset = oc.get(index)[0];
    out_data[out_offset] = std::conj(in_data[in_offset]);
  }
}

// In real-to-complex transform, cuFFT only fills half of the values due to
// conjugate symmetry. See native/SpectralUtils.h for more details.
// The following function fills in the other half with symmetry in
// case of real-to-complex transform with onesided=False flag.
// See NOTE [ Fourier Transform Conjugate Symmetry ] in native/SpectralOpsUtils.h.

// input should be a tensor of same size as full (twosided)
// signals, but only contains half (onesided) of the values.
// This function modifies inplace.
void _fft_fill_with_conjugate_symmetry_cuda_(
    ScalarType dtype, IntArrayRef mirror_dims, IntArrayRef signal_half_sizes,
    IntArrayRef in_strides, const void * in_data,
    IntArrayRef out_strides, void * out_data) {
  // Do the actual conjugate mirroring.
  // TODO: consider adding a 32bit indexed kernel for improved performance
  auto* in_strides_ptr = in_strides.data();
  const int ndim = in_strides.size();
  const int64_t element_size = scalarTypeToTypeMeta(dtype).itemsize();
  OffsetCalculator<1, int64_t> input_offset_calculator(
      ndim, signal_half_sizes.data(), &in_strides_ptr, &element_size);
  HermitianSymmetryOffsetCalculator<int64_t> output_offset_calculator(
      signal_half_sizes, out_strides, mirror_dims, element_size);

  const auto numel = at::prod_intlist(signal_half_sizes);
  AT_DISPATCH_COMPLEX_TYPES(dtype, "_fft_fill_with_conjugate_symmetry", [&] {
        using namespace cuda::detail;
        _fft_conjugate_copy_kernel<<<
          GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
              numel,
              static_cast<scalar_t*>(out_data),
              static_cast<const scalar_t*>(in_data),
              input_offset_calculator,
              output_offset_calculator);
      });
}

REGISTER_DISPATCH(fft_fill_with_conjugate_symmetry_stub, &_fft_fill_with_conjugate_symmetry_cuda_);

// NOTE [ cuFFT Embedded Strides ]
//
// cuFFT supports a subset of arbitrary strides via their "advanced data layout"
// option (http://docs.nvidia.com/cuda/cufft/index.html#advanced-data-layout).
// Specifically, these are tensors that can be viewed as subtensors resulted
// from slicing a larger contiguous tensors. For such input tensors, let the
// sizes of the enclosing tensor be `inembed`, and we can have in 3d case:
//
//     input[x, y, z] = input[((x * inembed[1] + y) * inembed[2] + z)]
//
// Above is the simplified formula ignoring the batch dimension. In fact, the
// last dimension of the enclosing tensor doesn't have to be contiguous, i.e.,
// it can be greater than 1. Then one can set the base stride for the enclosing
// tensor with `istride`. Then we have
//
//     input[x, y, z] = input[((x * inembed[1] + y) * inembed[2] + z) * istride]
//
// For example, consider
//
//     enclosing = torch.zeros(6, 8, 10)  # contiguous
//     input = enclosing[:4, 2:6, 6:]
//     input.size()                       # [ 4,  4,  4]
//     input.stride()                     # [80, 10,  1]
//     # inembed = [6, 8, 10]
//     input[2, 1, 3] = input[((2 * 8) + 1) * 10 + 3]   # using above formula
//                    = input[173]
//                    = input[2 * 80 + 1 * 10 + 1 * 3]  # using strides directly
//
// Generally, the embedded strides can be computed as
//
//     embed[i] = stride[i - 1] / stride[i].
//
// Note that the value of embed[0] isn't used to compute indices and doesn't
// matter.
//
// Contrary to advanced data layout, simple layout means that *embeds have
// unit-strides. In particular, unit-stride refers to that the input and output
// tensors being contiguous, and that the strides at the innermost signal
// dimension being unit (1) w.r.t. the corresponding data type.

static inline Tensor _run_cufft(
    const CuFFTConfig &config, Tensor& input, int64_t signal_ndim,
    bool complex_input, bool complex_output, bool inverse,
    IntArrayRef checked_signal_sizes, fft_norm_mode norm, bool onesided,
    IntArrayRef output_sizes, bool input_was_cloned
) {
  if (config.should_clone_input() && !input_was_cloned) {
    input = input.clone(at::MemoryFormat::Contiguous);
  }

  auto& plan = config.plan();
  auto& ctx = at::globalContext();

  // set output
  auto output = at::empty(output_sizes, input.options());

  // set to current stream
  CUFFT_CHECK(hipfftSetStream(plan, at::cuda::getCurrentCUDAStream()));

  auto ws = at::empty({ config.workspace_size() }, at::device(at::kCUDA).dtype(at::kByte));
  CUFFT_CHECK(hipfftSetWorkArea(plan, ws.data_ptr()));

  // run
#ifdef __HIP_PLATFORM_HCC__
  if (input.scalar_type() == ScalarType::Float) {
      if (complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecC2C(plan, static_cast<hipfftComplex*>(input.data_ptr()),
          static_cast<hipfftComplex*>(output.data_ptr()),
          inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD));
      } else if (complex_input && !complex_output) {
        CUFFT_CHECK(hipfftExecC2R(plan, static_cast<hipfftComplex*>(input.data_ptr()),
          static_cast<hipfftReal*>(output.data_ptr())));
      } else if (!complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecR2C(plan, static_cast<hipfftReal*>(input.data_ptr()),
          static_cast<hipfftComplex*>(output.data_ptr())));
      } else {
        AT_ERROR("hipFFT doesn't support r2r (float)");
      }
    } else if (input.scalar_type() == ScalarType::Double) {
      if (complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecZ2Z(plan, static_cast<hipfftDoubleComplex*>(input.data_ptr()),
          static_cast<hipfftDoubleComplex*>(output.data_ptr()),
          inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD));
      } else if (complex_input && !complex_output) {
        CUFFT_CHECK(hipfftExecZ2D(plan, static_cast<hipfftDoubleComplex*>(input.data_ptr()),
          static_cast<hipfftDoubleReal*>(output.data_ptr())));
      } else if (!complex_input && complex_output) {
        CUFFT_CHECK(hipfftExecD2Z(plan, static_cast<hipfftDoubleReal*>(input.data_ptr()),
          static_cast<hipfftDoubleComplex*>(output.data_ptr())));
      } else {
        AT_ERROR("hipFFT doesn't support r2r (double)");
      }
    } else {
      std::ostringstream ss;
      ss << "hipFFT doesn't support tensor of type: "
         << toString(input.scalar_type());
      AT_ERROR(ss.str());
    }
#else
  CUFFT_CHECK(hipfftXtExec(plan, input.data_ptr(), output.data_ptr(),
    inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD));
#endif

  // rescale if requested
  auto size_last_signal_dim = checked_signal_sizes[signal_ndim - 1];
  if (norm != fft_norm_mode::none) {
    auto signal_numel = at::prod_intlist(checked_signal_sizes);
    double scale_denom;
    if (norm == fft_norm_mode::by_root_n) {
      scale_denom = std::sqrt(static_cast<double>(signal_numel));
    } else {
      scale_denom = static_cast<double>(signal_numel);
    }
    if (!complex_input && complex_output && !onesided) {
      auto end_data_slice = infer_ft_real_to_complex_onesided_size(size_last_signal_dim);
      output.narrow(signal_ndim, 0, end_data_slice).div_(scale_denom);
    } else {
      output.div_(scale_denom);
    }
  }

  // if needed, fill out the other half using conjugate symmetry
  if (!complex_input && complex_output && !onesided) {
    DimVector signal_dims(signal_ndim);
    std::iota(signal_dims.begin(), signal_dims.end(), 1);
    auto out_as_complex = at::view_as_complex(output);
    at::native::_fft_fill_with_conjugate_symmetry_(out_as_complex, signal_dims);
  }
  return output;
}

// The cuFFT plan cache
// unique_ptr for nullability and to avoid reference invalidation on vector resize
static std::vector<std::unique_ptr<CuFFTParamsLRUCache>> plan_caches;
static std::mutex plan_caches_mutex;

static inline
CuFFTParamsLRUCache &cufft_get_plan_cache(int64_t device_index) {
  std::lock_guard<std::mutex> guard(plan_caches_mutex);

  AT_ASSERT(device_index >= 0);

  if (device_index >= plan_caches.size()) {
    plan_caches.resize(device_index + 1);
  }

  if (!plan_caches[device_index]) {
    plan_caches[device_index] = std::make_unique<CuFFTParamsLRUCache>();
  }

  return *plan_caches[device_index];
}


namespace detail {

int64_t cufft_get_plan_cache_max_size_impl(int64_t device_index) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_get_plan_cache_max_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).max_size();
}

void cufft_set_plan_cache_max_size_impl(int64_t device_index, int64_t max_size) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_set_plan_cache_max_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).resize(max_size);
}

int64_t cufft_get_plan_cache_size_impl(int64_t device_index) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_get_plan_cache_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).size();
}

void cufft_clear_plan_cache_impl(int64_t device_index) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_clear_plan_cache: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).clear();
}

} // namespace at::native::detail

// cuFFT
// Currently not utilizing multi GPUs so this can be potentially sped up.
Tensor _fft_cufft(const Tensor& self, int64_t signal_ndim,
                  bool complex_input, bool complex_output, bool inverse,
                  IntArrayRef checked_signal_sizes, int64_t normalization, bool onesided,
                  IntArrayRef output_sizes) {

  CuFFTParamsLRUCache& plan_cache = cufft_get_plan_cache(self.device().index());

  Tensor input = self;
  bool input_was_cloned = false;

  // Slice when twosided complex-to-real. This is not always needed because we
  // calculate the inembed. But it will benefit us in certain cases where we
  // clone the input tensor.
  //
  // See NOTE [ cuFFT Embedded Strides ].
  // See NOTE [ Fourier Transform Conjugate Symmetry ] in native/SpectralOpsUtils.h.
  if (complex_input && !complex_output && !onesided) {
    auto onesided_size = infer_ft_real_to_complex_onesided_size(checked_signal_sizes[signal_ndim - 1]);
    input = input.narrow(signal_ndim, 0, onesided_size);
  }

  // cuFFT requires input and output data pointers to complex type aligned.
  // Our newly allocated output tensor is always 512 bytes aligned so it is fine
  // (see kRoundSmall and kRoundLarge in THCCachingAllocator.cpp), but we do
  // need to check input tensor to make sure that it is not unaligned, e.g.,
  // from a slicing.
  auto complex_size_bytes = 2 * input.element_size();
  if (reinterpret_cast<std::uintptr_t>(input.data_ptr()) % complex_size_bytes != 0) {
    input = input.clone(at::MemoryFormat::Contiguous);
    input_was_cloned = true;
  }

  // Now that we have done error check and data_ptr checks, we delegate all
  // further cuFFT parameter computation and plan creation to the helper class
  // CuFFTConfig in CuFFTPlanCache.h.

  // If plan caching is enabled, we check the cache. Note that this accesses
  // plan_cache.max_size() and thus makes this function less functional.
  // However, integrating additional arguments into the "public" level c++ APIs,
  // e.g., irfft, is difficult as we have a long call sequence looking like
  //   irfft --> _fft --> _fft_with_size --dispatching-to-> _fft_cufft

  // This read is not locked for perf reason. Shouldn't matter too much because
  // we check again after acquiring the lock.
  if (plan_cache.max_size() > 0) {
    CuFFTParams params;
    setCuFFTParams(&params, input, signal_ndim, complex_input,
      complex_output, checked_signal_sizes, onesided);
    std::lock_guard<std::mutex> guard(plan_cache.mutex);
    if (plan_cache.max_size() > 0) {  // check again after acquiring the lock
      const CuFFTConfig &config = plan_cache.try_emplace_value(std::move(params),
                                             input, signal_ndim, complex_input,
                                             complex_output, checked_signal_sizes,
                                             onesided, output_sizes);
      return _run_cufft(config, input, signal_ndim, complex_input,
                        complex_output, inverse, checked_signal_sizes,
                        static_cast<fft_norm_mode>(normalization),
                        onesided, output_sizes, input_was_cloned);
    }
  }
  CuFFTConfig config(input, signal_ndim, complex_input, complex_output,
                     checked_signal_sizes, onesided, output_sizes);
  return _run_cufft(config, input, signal_ndim, complex_input,
                    complex_output, inverse, checked_signal_sizes,
                    static_cast<fft_norm_mode>(normalization),
                    onesided, output_sizes, input_was_cloned);
}

}} // at::native
