#include "hip/hip_runtime.h"
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

void abs_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Bool, iter.dtype(), "abs_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::abs(a);
    });
  });
}

REGISTER_DISPATCH(abs_stub, &abs_kernel_cuda);

}} // namespace at::native
