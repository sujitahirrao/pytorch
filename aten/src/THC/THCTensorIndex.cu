#include "hip/hip_runtime.h"
#include <THC/THC.h>
#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <THC/THCTensorCopy.h>
#include <TH/THHalf.h>
#include <THC/THCApply.cuh>
#include <THC/THCReduce.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <THC/THCThrustAllocator.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <algorithm> // for std::min
#include <c10/macros/Macros.h>
#include <ATen/WrapDimUtils.h>

template <int Dims, typename T, typename IndexType>
__device__ __forceinline__ IndexType indexToOffset(
    const TensorInfo<T, IndexType>& info,
    int64_t index,
    IndexType size)
{
  IndexType linearIndex = static_cast<IndexType>(index);
  CUDA_KERNEL_ASSERT(linearIndex < size && linearIndex >= -size);
  if (linearIndex < 0) {
    linearIndex += size;
  }
  return IndexToOffset<T, IndexType, Dims>::get(linearIndex, info);
}

struct WrapIndexOp {
  WrapIndexOp(int64_t size) : size(size) {}

  __device__ __forceinline__ void operator()(int64_t* out, int64_t* in) {
    auto idx = *in;
    CUDA_KERNEL_ASSERT(idx < size && idx >= -size);
    *out = idx < 0 ? idx + size : idx;
  }

  int64_t size;
};

template <typename T, typename IndexType, int Dims>
struct TensorPutOp {
  TensorPutOp(TensorInfo<T, IndexType> info, IndexType numel, int64_t*, int64_t*)
    : info(info), numel(numel) {}

  __device__ __forceinline__ void operator()(T* value, int64_t* index) {
    auto offset = indexToOffset<Dims>(info, *index, numel);
    info.data[offset] = *value;
  }

  const TensorInfo<T, IndexType> info;
  IndexType numel;
};

template <typename T, typename IndexType, int Dims>
struct TensorPutAccumulateOp {
  TensorPutAccumulateOp(TensorInfo<T, IndexType> info, IndexType numel, int64_t* start, int64_t* end)
    : info(info), numel(numel), start(start), end(end) {}

  __device__ __forceinline__ void operator()(T* value, int64_t* index) {
    if (index == start || *index != *(index - 1)) {
      int64_t linear_index = *index;
      auto offset = indexToOffset<Dims>(info, linear_index, numel);
      do {
        info.data[offset] = THCNumerics<T>::add(info.data[offset], *value);
        index++;
        value++;
      } while (index != end && *index == linear_index);
    }
  }

  const TensorInfo<T, IndexType> info;
  IndexType numel;
  int64_t* start;
  int64_t* end;
};


template<typename IndexType, typename T, template<class, class, int> class Op, typename TensorType>
void dispatchTakePutImpl(THCState *state, TensorType *a, TensorType *b, THCudaLongTensor *index) {
  // These are only valid if index is contiguous
  auto start = THCudaLongTensor_data(state, index);
  auto end = start + THCudaLongTensor_numel(state, index);

  auto aInfo = getTensorInfo<T, TensorType, IndexType>(state, a);
  aInfo.collapseDims();
  auto numel = THCTensor_nElement(state, a);
  if (aInfo.isContiguous()) {
    auto op = Op<T, IndexType, -2>(aInfo, numel, start, end);
    THC_pointwiseApply2<T, int64_t>(state, b, index, op);
  } else {
    auto op = Op<T, IndexType, -1>(aInfo, numel, start, end);
    THC_pointwiseApply2<T, int64_t>(state, b, index, op);
  }
}

template<typename T, template<class, class, int> class Op, typename TensorType>
void dispatchTakePut(THCState *state, TensorType *a, TensorType *b, THCudaLongTensor *index) {
  if (THCTensor_canUse32BitIndexMath(state, a, INT_MAX)) {
    dispatchTakePutImpl<int32_t, T, Op>(state, a, b, index);
  } else {
    dispatchTakePutImpl<int64_t, T, Op>(state, a, b, index);
  }
}

#include <THC/generic/THCTensorIndex.cu>
#include <THC/THCGenerateAllTypes.h>

#include <THC/generic/THCTensorIndex.cu>
#include <THC/THCGenerateBoolType.h>

#include <THC/generic/THCTensorIndex.cu>
#include <THC/THCGenerateBFloat16Type.h>
